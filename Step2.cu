#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

// Step 2
__global__
void initialMatching(bool* coveredRows, bool* coveredCols, int32_t* rowOfStarAtColumn, int32_t* colOfStarAtRow, int32_t* zeroes, int32_t numCols) {
    int32_t thread = threadIdx.x;
    int32_t block = blockIdx.x;
    int32_t flag = 0, pos, r, c;

    while (flag != 1) {
        for(int32_t i = 0; i < numberOfZerosInBlock[block]; i++) {
            pos = zeros[i * blockDim.x + thread]
            r = pos % numCols;
            c = pos / numCols;
            flag = 1;
            if (coveredRows[r] && coveredCols[c])
                if (!atomicExch(&coveredRows[r], true))
                    if (!atomicExch(&coveredCols[c], true)) {
                        rowOfStarAtColumn[r] = c;
                        colOfStarAtRow[c] = r;
                    } else {
                        coveredRows[r] = false;
                        flag = 0;
                    }
        }
    }
    __syncthreads()
}

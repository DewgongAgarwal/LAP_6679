#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

//Step 3

__global__
void searchInitialization(bool* coveredRows, bool* coveredCols, int32_t* numStarredZeroes) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    coveredRows[i] = false;
    coveredCols[i] = false;
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *numStarredZeroes = 0;
    }
}

__global__
void searchTermination(bool* rowOfStarAtColumn, bool* coveredCols, int32_t* numStarredZeroes) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (rowOfStarAtColumn[i] != -1) {
        coveredCols[i] = true;
        atomicAdd(&numStarredZeroes, 1);
    }
}
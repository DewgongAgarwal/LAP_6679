#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

//Step 4

void alternatingGraphPathSearch(bool* coveredRows, bool* coveredCols, int32_t* colOfPrimeAtRow, int32_t* rowOfStarAtColumn) {
    volatile bool* copyCoveredRows = coveredRows;
    volatile bool* copyCoveredCols = coveredCols;

    bool zeroFound = false, goToStep5 = false, repeatKernel = false;

    do {
        __syncthreads();
        zeroFound = false;
        __syncthreads();

        for(int32_t i = 0; i < numberOfZerosInBlock[block]; i++) {
            pos = zeros[i * blockDim.x + thread]
            r = pos % numCols;
            c = pos / numCols;
            if (!copyCoveredCols[c] && !copyCoveredRows[r]) {
                colOfPrimeAtRow[r] = c;
                repeatKernel = true;
                if (rowOfStarAtColumn[c] != -1) {
                    copyCoveredRows[r] = true;
                    __threadfence();
                    copyCoveredCols[c] = false;
                } else {
                    goToStep5 = true;
                }
            }
        }
        __syncthreads();
        if (goToStep5) {
            break;
        }
    } while (zeroFound);
    __syncthreads();
}
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

// Step 5a
__global__
void removeOverlappingPaths(int32_t* colOfPrimeAtRow, int32_t* colOfStarAtRow, int32_t* rowOfStarAtCol, int32_t* rowOfGreenAtCol) {
    int32_t = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t col = colOfPrimeAtRow[i];
    int32_t starCol = colOfStarAtRow[i];
    int32_t row = -1;
    if (col != -1 && starCol != -1) {
        rowOfGreenAtCol[col] = i;
        while(rowOfStarAtCol[col] != -1) {
            row = rowOfStarAtCol[col];
            col = colOfPrimeAtRow[col];
            rowOfGreenAtCol[col] = row;
        }
    }
}

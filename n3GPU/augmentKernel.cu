#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void augmentKernel(int32_t* aug, int32_t* Sr, int32_t* Sc, int32_t* Ar, 
    int32_t* Ac, int32_t size) {
    
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t rcur = i;
    int32_t ccur = -1;
    if (i < size && aug[i] != 0) {
        while (rcur != -1) {
            ccur = Sr[rcur];
            Ar[rcur] = ccur;
            Ac[ccur] = rcur;
            rcur = Sc[ccur];
        }
    }
    
}

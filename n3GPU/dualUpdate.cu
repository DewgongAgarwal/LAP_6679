#include <stdint.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void minKernel(double* min, double* newMin, int32_t size) {
    __shared__ double partial[32];
    int32_t index = threadIdx.x;
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    double x = 0;
    double y = 0;
    if (i < size) {
        int32_t temp = min[i];
        partial[threadIdx.x] = (temp > -0.0001) * temp + (1 - temp > -0.0001) * INT_MAX; 
    } else {
        int32_t temp = min[0];
        partial[threadIdx.x] = (temp > -0.0001) * temp + (1 - temp > -0.0001) * INT_MAX;
    }
    for(int32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();

        if (index < stride) {
            x = partial[index];
            y = partial[index + stride];
            partial[index] = (x < y) * x + (1 - x < y) * y;
        }
    }
    if(index == 0) {
        newMin[blockIdx.x] = partial[index];
    }
}

__global__ void dualUpdate(double theta, int32_t* Vr, int32_t* Vc, int32_t* Pc, double* Dr, double* Dc, double* slack, int32_t size, int32_t* frontier, int32_t* countNewFrontier) {
    int32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k  < size) {
        double Dr1 = Dr[k];
        double Dc1 = Dc[k];
        double slacks = slack[k];
        double tempSlack = slacks;
        Dr1 += (-2 * Vr[k] + 1) * theta / 2.0;
        Dc1 += (-2 * Vc[k] + 1) * theta / 2.0;

        slacks -= (slacks > 0) * theta;
        frontier[Pc[k]] = frontier[Pc[k]] || (tempSlack > 0 && slacks <= 0.0001);
        *countNewFrontier = *countNewFrontier || (tempSlack > 0 && slacks <= 0.0001);

        Dr[k] = Dr1;
        Dc[k] = Dc1;
        slack[k] = slacks;
    }
}

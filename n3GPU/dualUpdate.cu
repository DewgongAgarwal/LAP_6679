#include <stdint.h>
#include <hip/hip_runtime.h>
#include <limits.h>

__global__ void minKernel(double* min, double* newMin, int32_t size) {
    __shared__ double partial[512];
    int32_t index = threadIdx.x;
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    double x = 0;
    double y = 0;
    if (i < size) {
        int32_t temp = min[i];
        if (temp > 0) {
            partial[threadIdx.x] = temp; 
        } else {
            partial[threadIdx.x] = (double)INT_MAX;
        }
    } else {
        partial[threadIdx.x] = (double)INT_MAX;
    }
    for(int32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();

        if (index < stride) {
            x = partial[index];
            y = partial[index + stride];
            partial[index] = fmin(x, y);
        }
    }
    if(index == 0) {
        newMin[blockIdx.x] = partial[index];
    }
}

__global__ void dualUpdate(double theta, int32_t* Vr, int32_t* Vc, int32_t* Pc,
    double* Dr, double* Dc, double* slack, int32_t size, int32_t* frontier, 
    int32_t* countNewFrontier, int32_t* lengthOfFrontier) {
    
    int32_t k = blockIdx.x * blockDim.x + threadIdx.x;
    if (k  < size) {

        Dc[k] = fma(0.5 - Vc[k], theta, Dc[k]);
        Dr[k] = fma(0.5 - Vr[k], theta, Dr[k]);
        double slacksCopy = slack[k];
        if (slacksCopy > 0.0) {
            slacksCopy = fma(-1.0, theta, slacksCopy);
            if (slacksCopy == 0.0) {
                *countNewFrontier = 1;
                int32_t current = atomicAdd(lengthOfFrontier, 1);
                frontier[current] = Pc[k];
            }
            slack[k] = slacksCopy;
        }
    }
}

#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void frontierInit(int32_t* frontier, int32_t* newFrontier, 
    int32_t* visited, int32_t* reverse, int32_t* Ar, int32_t workers,
    int32_t* lengthOfFrontier) {
    
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < workers && Ar[i] == -1) {
        int32_t current = atomicAdd(lengthOfFrontier, 1);
        frontier[current] = i;
    }
}

__global__ void visitedInit(int32_t* visited, int32_t* frontier, int32_t workers, int32_t* lengthOfFrontier) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < *lengthOfFrontier) {
        visited[frontier[i]] = 1;
    }
}

__global__ void forward(int32_t* frontier, int32_t* costs, double* Dr, double* Dc, 
                        int32_t* Ac, int32_t* Vr, int32_t* Vc, double* slack, 
                        int32_t* Pr, int32_t* Pc, int32_t workers, int32_t* newFrontier, 
                        int32_t* reverse, int32_t* countReverse, 
                        int32_t* visited, int32_t* lengthOfFrontier, int32_t* lengthOfNewFrontier) {
    
    int32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    double eps = 1e-6;
    if (j < workers && Vc[j] == 0) {
        int32_t inew = Ac[j];
        double DcCopy = Dc[j];
        double slacksCopy = slack[j];
        bool inReverse = false, inNewFrontier = false;
        for(int32_t i1 = 0; i1 < *lengthOfFrontier; i1++) {
            int32_t i = frontier[i1];
            double DrCopy = Dr[i];
            int32_t costsCopy = costs[j * workers + i];
            if (slacksCopy > costsCopy - DrCopy - DcCopy) {
                slacksCopy = costsCopy - DrCopy - DcCopy;
                Pc[j] = i;
            }
            if (slacksCopy < eps && slacksCopy > -eps) {
                inNewFrontier = (inew != -1);
                inReverse = (inew == -1);
            }
        }
        if (inNewFrontier) {
            Pr[inew] = j;
            Vr[inew] = 0;
            Vc[j] = 1;
            if (!visited[inew]) {
                int32_t offset = atomicAdd(lengthOfNewFrontier, 1);
                newFrontier[offset] = inew;
            }
        }
        if (inReverse) {
            *countReverse = 1;
            reverse[j] = 1;
        }
        slack[j] = slacksCopy;
    }
}
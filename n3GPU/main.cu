#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "step6.cu"
#include "step5.cu"
#include "step4.cu"
#include "step3.cu"
#include "step2.cu"
#include "step1.cu"
#include <stdbool.h>

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;


Timer timer;
Timer timer1;
Timer timer2;
Timer timer3;
Timer timer4;
Timer timer5;
Timer timer6;
hipError_t cuda_ret;

#define FATAL(_msg, ...) \
    do {\
      fprintf(stderr, "[%s:%d] " _msg "\n", __FILE__, __LINE__, ##__VA_ARGS__); \
      exit(-1);								\
    } while(0)

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

double elapsedTime(Timer timer) {
    return ((double) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

__host__ void printArray(int32_t* a, int32_t size) {
      for(int32_t i = 0; i < size; i++) {
          printf("%d, ", a[i]);
      }
      printf("\n");
  }

  __host__ void printArray(double* a, int32_t size) {
    for(int32_t i = 0; i < size; i++) {
        printf("%lf, ", a[i]);
    }
    printf("\n");
  }
  
  __host__ void flatten(int32_t** costs, int32_t workers, int32_t* costsRowwise) {
      for(int32_t i = 0; i < workers; i++) {
          for(int32_t j = 0; j < workers; j++) {
              costsRowwise[i * workers + j] = costs[i][j];
          }
      }
  }

  __global__ void initialize(int32_t* a, int32_t initialValue, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        a[i] = initialValue;
    }
}

__global__ void initialize(double* a, double initialValue, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        a[i] = initialValue;
    }
}

  __host__ void declareAndInitialize(int32_t** a, int32_t initialValue, int32_t initialValueGiven, int32_t size) {
      cuda_ret = hipMalloc((void**) a, sizeof(int32_t) * size);
      if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
      if(initialValueGiven == 1) {
            int32_t threads = 256;
            int32_t blocks = (size + threads - 1) / threads;
            initialize<<<blocks, threads>>>(*a, initialValue, size);
      }
  }

    __host__ void declareAndInitialize(double** a, double initialValue, int32_t initialValueGiven, int32_t size) {
        cuda_ret = hipMalloc((void**) a, sizeof(double) * size);
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
        if(initialValueGiven == 1) {
            int32_t threads = 256;
            int32_t blocks = (size + threads - 1) / threads;
            initialize<<<blocks, threads>>>(*a, initialValue, size);
      }
    }

  void generateCosts(int32_t** costs, int32_t workers) {
    for(int32_t i = 0; i < workers; i++) {
        for(int32_t j = 0; j < workers; j++) {
            costs[i][j] = rand() % (workers / 10 + 1);
        }
    }
}
  
  int n3Hungarian(int32_t workers) {
    //srand(0);
    double step1Time = 0.0, step2Time = 0.0, step3Time = 0.0, step4Time = 0.0, step5Time = 0.0, step6Time = 0.0;
    int32_t** costs = (int32_t**)calloc(workers, sizeof(int32_t*));
    for(int32_t i = 0; i < workers; i++) {
        costs[i] = (int32_t*)calloc(workers, sizeof(int32_t));
    }
    generateCosts(costs, workers);
      
    startTime(&timer);
    int32_t* costsRowwise_h = (int32_t*)calloc(workers * workers, sizeof(int32_t));
    int32_t* answers = (int32_t*)calloc(workers, sizeof(int32_t));
    int32_t* repeat = (int32_t*)calloc(1, sizeof(int32_t));
    flatten(costs, workers, costsRowwise_h);
    printf("Workers: %d\n", workers);

    int32_t* costsRowwise_d;
    int32_t* costsColwise_d;
    double* Dr;
    double* Dc;
    int32_t* Ar;
    int32_t* Ac;
    int32_t* Vr;
    int32_t* Vc;
    int32_t* Pc;
    int32_t* Pr;
    int32_t* Sc;
    int32_t* Sr;
    double* slacks;
    double* newSlack;
    double* newSlack2;
    int32_t* frontier;
    int32_t* newFrontier;
    int32_t* reverse;
    int32_t* countNewFrontier;
    int32_t* countReverse;
    int32_t* augment;
    int32_t* visited;
    int32_t* countsSum;
    int32_t* lengthOfFrontier;
    int32_t* lengthOfNewFrontier;


    int32_t threads = 32;
    int32_t blocks = (workers + threads - 1) / threads;

    cuda_ret = hipMalloc((void**) &countNewFrontier, sizeof(int32_t));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(countNewFrontier, 0, sizeof(int32_t));
    cuda_ret = hipMalloc((void**) &countReverse, sizeof(int32_t));
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
    hipMemset(countReverse, 0, sizeof(int32_t));

    declareAndInitialize(&costsRowwise_d, 0, 0, workers * workers);
    declareAndInitialize(&costsColwise_d, 0, 0, workers * workers);

    declareAndInitialize(&Dr, 0, 0, workers);
    declareAndInitialize(&Dc, 0, 1, workers);

    declareAndInitialize(&Ar, -1, 1, workers);
    declareAndInitialize(&Ac, -1, 1, workers);

    declareAndInitialize(&Vr, 0, 0, workers);
    declareAndInitialize(&Vc, 0, 0, workers);

    declareAndInitialize(&Pc, -1, 0, workers);
    declareAndInitialize(&Pr, -1, 0, workers);

    declareAndInitialize(&Sc, -1, 0, workers);
    declareAndInitialize(&Sr, -1, 0, workers);

    declareAndInitialize(&slacks, 0, 0, workers);
    declareAndInitialize(&newSlack, 0, 0, workers);
    declareAndInitialize(&newSlack2, 0, 0, workers);

    declareAndInitialize(&frontier, 1, 0, workers);
    declareAndInitialize(&newFrontier, 0, 0, workers);  

    declareAndInitialize(&reverse, 0, 0, workers);
    declareAndInitialize(&augment, 0, 0, workers);

    declareAndInitialize(&visited, 0, 0, workers);

    declareAndInitialize(&countsSum, 0, 0, 1);

    declareAndInitialize(&lengthOfFrontier, 0, 1, 1);
    declareAndInitialize(&lengthOfNewFrontier, 0, 1, 1);
    
    bool done = false;
    int32_t performOptimalityCheck = 1;
    int32_t* matchCount = (int32_t*) calloc(1, sizeof(int32_t));
    int32_t* goToStep4 = (int32_t*)calloc(1, sizeof(int32_t));
    startTime(&timer1);
    step1(costsRowwise_d, costsColwise_d, costsRowwise_h, Dr, Dc, workers, blocks, threads);
    stopTime(&timer1);
    step1Time += elapsedTime(timer1);
    int32_t sum = 0;
    do {
        if (performOptimalityCheck) {
            startTime(&timer2);
            done = step2(Ar, Vr, countsSum, workers, Pc, Sr, Pr, Sc, Vc, slacks, matchCount, blocks, threads);
            stopTime(&timer2);
            step2Time += elapsedTime(timer2);
        }
        if (done) {
            printf("Optimal solution found\n");
            hipMemcpy(answers, Ar, workers * sizeof(int32_t), hipMemcpyDeviceToHost);
            for(int i = 0; i < workers; i++) {
                sum += costs[i][answers[i]];
            }
            printf("\n");
            printf("Cost: %d\n", sum);
            break;
        } 
        *goToStep4 = 0;
        startTime(&timer3);
        step3(frontier, costsColwise_d, Dr, Dc, Ar, Ac, Vr, Vc, slacks, Pr, Pc,
                workers, newFrontier, repeat, reverse, countNewFrontier, countReverse,
                visited, goToStep4, blocks, threads, performOptimalityCheck, 
                lengthOfFrontier, lengthOfNewFrontier);
        stopTime(&timer3);
        step3Time += elapsedTime(timer3);
        if(*goToStep4) {
            startTime(&timer4);
            step4(reverse, Pr, Pc, Sr, Sc, augment, workers, blocks, threads);
            stopTime(&timer4);
            step4Time += elapsedTime(timer4);
            startTime(&timer5);
            step5(augment, Sr, Sc, Ar, Ac, workers, blocks, threads);
            performOptimalityCheck = 1;
            stopTime(&timer5);
            step5Time += elapsedTime(timer5);
        } else {
            startTime(&timer6);
            step6(slacks, newSlack, newSlack2, workers, Vr, Vc, Pc, Dr, Dc, 
                frontier, countNewFrontier, blocks, threads, lengthOfFrontier);
            hipMemcpy(&performOptimalityCheck, countNewFrontier, sizeof(int32_t), hipMemcpyDeviceToHost);
            performOptimalityCheck = 1 - performOptimalityCheck;
            stopTime(&timer6);
            step6Time += elapsedTime(timer6);
        }
    } while(1);

    hipFree(costsRowwise_d);
    hipFree(costsColwise_d);
    hipFree(Dr);
    hipFree(Dc);
    hipFree(Vr);
    hipFree(Vc);
    hipFree(Ar);
    hipFree(Ac);
    hipFree(Pc);
    hipFree(Pr);
    hipFree(Sc);
    hipFree(Sr);
    hipFree(slacks);
    hipFree(newSlack);
    hipFree(newSlack2);
    hipFree(frontier);
    hipFree(newFrontier);
    hipFree(reverse);
    hipFree(countNewFrontier);
    hipFree(countReverse);
    hipFree(augment);
    hipFree(visited);
    hipFree(countsSum);
    
    for(int32_t i = 0; i < workers; i++) {
        free(costs[i]);
    }
    printArray(matchCount, 1);
    stopTime(&timer); printf("%f s\n", elapsedTime(timer));
    printf("Step 1: %f s\n", step1Time);
    printf("Step 2: %f s\n", step2Time);
    printf("Step 3: %f s\n", step3Time);
    printf("Step 4: %f s\n", step4Time);
    printf("Step 5: %f s\n", step5Time);
    printf("Step 6: %f s\n", step6Time);
    free(costs);
    free(costsRowwise_h);
    free(matchCount);
    free(goToStep4);
    free(answers);
    free(repeat);
    return 0;
  }
  
int main() {
    for(int32_t i = 5; i < 15; i++) {
        n3Hungarian(1 << i);
        printf("\n\n");
    }
}

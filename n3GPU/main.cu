#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "rowReduction.cu"
#include "optimalityCheck.cu"
#include "passKernel.cu"
#include "augmentKernel.cu"
#include "dualUpdate.cu"
#include <sys/time.h>

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;


Timer timer;
hipError_t cuda_ret;

#define FATAL(_msg, ...) \
    do {\
      fprintf(stderr, "[%s:%d] " _msg "\n", __FILE__, __LINE__, ##__VA_ARGS__); \
      exit(-1);								\
    } while(0)

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

__host__ void printArray(int32_t* a, int32_t size) {
      for(int32_t i = 0; i < size; i++) {
          printf("%d, ", a[i]);
      }
      printf("\n");
  }

  __host__ void printArray(double* a, int32_t size) {
    for(int32_t i = 0; i < size; i++) {
        printf("%lf, ", a[i]);
    }
    printf("\n");
  }
  
  __global__ void frontierInit(int32_t* frontier, int32_t* Ar, int32_t workers) {
        int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
        if(i < workers) {
            frontier[i] = Ar[i] == -1;
        }
  }
  
  __global__ void visitedUpdate(int32_t* visited, int32_t* frontier, int32_t workers) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < workers) {
        visited[i] = visited[i] || frontier[i];
    }
}

  __host__ void generateCosts(int32_t** costs, int32_t workers, int32_t limit) {
      for(int32_t i = 0; i < workers; i++) {
          for(int32_t j = 0; j < workers; j++) {
              costs[i][j] = rand() % limit;
          }
      }
  }
  
  __host__ void flatten(int32_t** costs, int32_t workers, int32_t* costsRowwise, int32_t* costsColWise) {
      for(int32_t i = 0; i < workers; i++) {
          for(int32_t j = 0; j < workers; j++) {
              costsRowwise[i * workers + j] = costs[i][j];
              costsColWise[i * workers + j] = costs[j][i];
          }
      }
  }

  __global__ void initialize(int32_t* a, int32_t initialValue, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        a[i] = initialValue;
    }
}

__global__ void initialize(double* a, double initialValue, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        a[i] = initialValue;
    }
}

__global__ void copyKernel(int32_t* dest, int32_t* src, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        dest[i] = src[i];
    }
}

  __host__ void declareAndInitialize(int32_t** a, int32_t initialValue, int32_t initialValueGiven, int32_t size) {
      cuda_ret = hipMalloc((void**) a, sizeof(int32_t) * size);
      if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
      if(initialValueGiven == 1) {
            int32_t threads = 32;
            int32_t blocks = (size + threads - 1) / threads;
            initialize<<<blocks, threads>>>(*a, initialValue, size);
      }
  }

    __host__ void declareAndInitialize(double** a, double initialValue, int32_t initialValueGiven, int32_t size) {
        cuda_ret = hipMalloc((void**) a, sizeof(double) * size);
        if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
        if(initialValueGiven == 1) {
            int32_t threads = 32;
            int32_t blocks = (size + threads - 1) / threads;
            initialize<<<blocks, threads>>>(*a, initialValue, size);
      }
    }

  __host__ void optimalityCheck(int32_t* countsSum, int32_t* newCountSums, int32_t* Ar, int32_t* Vr, double* slacks, int32_t* matchCount, int32_t workers) {
        int32_t threads = 32;
        int32_t blocks = (workers + threads - 1) / threads;
        int32_t flag = -1;
        optimal<<<blocks, threads>>>(Ar, Vr, countsSum, workers);
        int32_t lengthOfCountSum = blocks;

        while(lengthOfCountSum > 1) {
            blocks = (lengthOfCountSum + threads - 1) / threads;
            sumKernel<<<blocks, threads>>>(countsSum, newCountSums, lengthOfCountSum);
            lengthOfCountSum /= threads;
            flag = 0;
            if (lengthOfCountSum > 1) {
                blocks = (lengthOfCountSum + threads - 1) / threads;
                sumKernel<<<blocks, threads>>>(newCountSums, countsSum, lengthOfCountSum);
                lengthOfCountSum /= threads;
                flag = 1;
            }
        }
        if (flag * flag == 1) {
            cuda_ret = hipMemcpy(matchCount, countsSum, sizeof(int32_t), hipMemcpyDeviceToHost);
        } else {
            cuda_ret = hipMemcpy(matchCount, newCountSums, sizeof(int32_t), hipMemcpyDeviceToHost);
        }
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the host for matchcount");
    }   

  __host__ void getMinSlack(double* slack, double* newSlack, double* newSlack2, int32_t workers, double* minSlack) {
        int32_t threads = 32;
        int32_t blocks = (workers + threads - 1) / threads;
        int32_t flag = -1;
        int32_t lengthOfCountSum = workers;
        double* slack2 = (double*) calloc(workers, sizeof(double));
        minKernel<<<blocks, threads>>>(slack, newSlack, lengthOfCountSum);
        hipDeviceSynchronize();
        hipMemcpy(slack2, newSlack, sizeof(double) * workers, hipMemcpyDeviceToHost);
        lengthOfCountSum = blocks;
        while(lengthOfCountSum > 1) {
            blocks = (lengthOfCountSum + threads - 1) / threads;
            minKernel<<<blocks, threads>>>(newSlack, newSlack2, lengthOfCountSum);
            lengthOfCountSum /= threads;
            flag = 0;
            if (lengthOfCountSum > 1) {
                blocks = (lengthOfCountSum + threads - 1) / threads;
                minKernel<<<blocks, threads>>>(newSlack2, newSlack, lengthOfCountSum);
                lengthOfCountSum /= threads;
                flag = 1;
            }
        }
        free(slack2);
        if (flag * flag == 1) {
            cuda_ret = hipMemcpy(minSlack, newSlack, sizeof(double), hipMemcpyDeviceToHost);
        } else {
            cuda_ret = hipMemcpy(minSlack, newSlack2, sizeof(double), hipMemcpyDeviceToHost);
        }
        if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the host for min-Slack");
  }

  void generateCosts(int32_t** costs, int32_t workers) {
    for(int32_t i = 0; i < workers; i++) {
        for(int32_t j = 0; j < workers; j++) {
            costs[i][j] = rand() % workers;
        }
    }
}
  
  int main() {
      int32_t workers = 8192 * 2;
    //   int32_t array[25] = {9, 22, 58, 11, 19, 43, 78, 72, 50, 63, 41, 28, 91, 37, 45, 74, 42, 27, 49, 39, 36, 11, 57, 22, 25};
      int32_t** costs = (int32_t**)calloc(workers, sizeof(int32_t*));
      for(int32_t i = 0; i < workers; i++) {
          costs[i] = (int32_t*)calloc(workers, sizeof(int32_t));
        //   for (int32_t j = 0; j < workers; j++) {
        //       costs[i][j] = array[i * workers + j];
        //   }
        }
        generateCosts(costs, workers);
      
        startTime(&timer);
      int32_t* costsRowwise_h = (int32_t*)calloc(workers * workers, sizeof(int32_t));
      int32_t* costsColwise_h = (int32_t*)calloc(workers * workers, sizeof(int32_t));
      flatten(costs, workers, costsRowwise_h, costsColwise_h);
      printf("Workers: %d\n", workers);
      // declareAndInitialize on device
      int32_t* costsRowwise_d;
      int32_t* costsColwise_d;
      double* Dr;
      double* Dc;
      int32_t* Ar;
      int32_t* Ac;
      int32_t* Vr;
      int32_t* Vc;
      int32_t* Pc;
      int32_t* Pr;
      int32_t* Sc;
      int32_t* Sr;
      double* slacks;
      double* newSlack;
      double* newSlack2;
      int32_t* frontier;
      int32_t* newFrontier;
      int32_t* reverse;
      int32_t* countNewFrontier;
      int32_t* countReverse;
      int32_t* augment;
      int32_t* visited;
      int32_t* countsSum;
      int32_t* newCountSums;


      int32_t threads = 32;
      int32_t blocks = (workers + threads - 1) / threads;

      cuda_ret = hipMalloc((void**) &countNewFrontier, sizeof(int32_t));
      if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
      hipMemset(countNewFrontier, 0, sizeof(int32_t));
      cuda_ret = hipMalloc((void**) &countReverse, sizeof(int32_t));
      if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
      hipMemset(countReverse, 0, sizeof(int32_t));

      declareAndInitialize(&costsRowwise_d, 0, 0, workers * workers);
      declareAndInitialize(&costsColwise_d, 0, 0, workers * workers);

      declareAndInitialize(&Dr, 0, 1, workers);
      declareAndInitialize(&Dc, 0, 1, workers);

      declareAndInitialize(&Ar, -1, 1, workers);
      declareAndInitialize(&Ac, -1, 1, workers);

      declareAndInitialize(&Vr, 0, 1, workers);
      declareAndInitialize(&Vc, 0, 1, workers);

      declareAndInitialize(&Pc, -1, 1, workers);
      declareAndInitialize(&Pr, -1, 1, workers);

      declareAndInitialize(&Sc, -1, 1, workers);
      declareAndInitialize(&Sr, -1, 1, workers);

      declareAndInitialize(&slacks, INT_MAX, 1, workers);
      declareAndInitialize(&newSlack, 0, 1, workers);
      declareAndInitialize(&newSlack2, 0, 1, workers);

      declareAndInitialize(&frontier, 1, 1, workers);
      declareAndInitialize(&newFrontier, 0, 1, workers);  

      declareAndInitialize(&reverse, 0, 1, workers);
      declareAndInitialize(&augment, 0, 1, workers);

      declareAndInitialize(&visited, 0, 1, workers);

      declareAndInitialize(&countsSum, 0, 1, blocks);
      declareAndInitialize(&newCountSums, 0, 1, blocks);

      // copy to device
      cuda_ret = hipMemcpy(costsRowwise_d, costsRowwise_h, sizeof(int32_t) * workers * workers, hipMemcpyHostToDevice);
      if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device for Row-wise");
      cuda_ret = hipMemcpy(costsColwise_d, costsColwise_h, sizeof(int32_t) * workers * workers, hipMemcpyHostToDevice);
      if(cuda_ret != hipSuccess) FATAL("Unable to copy memory to the device for Col-wise");
  
      findRowMin<<<blocks, threads>>>(costsRowwise_d, Dr, Dc, workers);
      hipDeviceSynchronize();
      findRowMin<<<blocks, threads>>>(costsColwise_d, Dc, Dr, workers);
      hipDeviceSynchronize();
      
      int32_t flag = 0;
      int32_t* matchCount = (int32_t*) calloc(1, sizeof(int32_t));
      int32_t* lengthOfFrontier_h = (int32_t *)calloc(1, sizeof(int32_t));
      int32_t* lengthOfReverse_h = (int32_t *)calloc(1, sizeof(int32_t));
      int32_t* Ar_h = (int32_t*)calloc(workers, sizeof(int32_t));
      double* thetaCopy = (double*)calloc(1, sizeof(double));

      *lengthOfFrontier_h = 1;
      do {
        hipMemset(Vc, 0, sizeof(int32_t) * workers);
        hipMemset(Vr, 0, sizeof(int32_t) * workers);
        initialize<<<blocks, threads>>>(Pc, -1, workers);
        initialize<<<blocks, threads>>>(Pr, -1, workers);
        initialize<<<blocks, threads>>>(Sr, -1, workers);
        initialize<<<blocks, threads>>>(Sc, -1, workers);
        hipDeviceSynchronize();
        optimalityCheck(countsSum, newCountSums, Ar, Vr, slacks, matchCount, workers);
        initialize<<<blocks, threads>>>(slacks, INT_MAX, workers);
        hipDeviceSynchronize();
        if (*matchCount == workers) {
            printf("Optimal solution found\n");
            hipMemcpy(Ar_h, Ar, sizeof(int32_t) * workers, hipMemcpyDeviceToHost);
            printArray(Ar_h, workers);
            break;
        } else {
            do {
                flag = 0;
                frontierInit<<<blocks, threads>>>(frontier, Ar, workers);
                hipDeviceSynchronize();
                hipMemset(visited, 0, sizeof(int32_t) * workers);
                hipDeviceSynchronize();
                while(*lengthOfFrontier_h != 0) {
                    hipMemset(countNewFrontier, 0, sizeof(int32_t));
                    hipMemset(countReverse, 0, sizeof(int32_t));
                    forward<<<blocks, threads>>>(frontier, costsColwise_d, Dr, Dc, Ac, Vr, Vc, slacks, Pr, Pc, workers, newFrontier, reverse, countNewFrontier, countReverse, visited);
                    hipDeviceSynchronize();

                    visitedUpdate<<<blocks, threads>>>(visited, frontier, workers);
                    hipDeviceSynchronize();

                    hipMemset(frontier, 0, sizeof(int32_t) * workers);
                    hipMemcpy(lengthOfFrontier_h, countNewFrontier, sizeof(int32_t), hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    if (*lengthOfFrontier_h == 0) {
                        flag = 1;
                        break;
                    }
                    hipMemset(countNewFrontier, 0, sizeof(int32_t));
                    hipMemset(countReverse, 0, sizeof(int32_t));
                    hipDeviceSynchronize();
                    forward<<<blocks, threads>>>(newFrontier, costsColwise_d, Dr, Dc, Ac, Vr, Vc, slacks, Pr, Pc, workers, frontier, reverse, countNewFrontier, countReverse, visited);
                    hipDeviceSynchronize();
                    visitedUpdate<<<blocks, threads>>>(visited, newFrontier, workers);
                    hipDeviceSynchronize();
                    hipMemset(newFrontier, 0, sizeof(int32_t) * workers);
                    hipDeviceSynchronize();
                    hipMemcpy(lengthOfFrontier_h, countNewFrontier, sizeof(int32_t), hipMemcpyDeviceToHost);
                    hipDeviceSynchronize();
                    flag = 0;
                }
                
                hipMemcpy(lengthOfReverse_h, countReverse, sizeof(int32_t), hipMemcpyDeviceToHost);
                hipDeviceSynchronize();
                if(*lengthOfReverse_h == 0) {
                    break;
                }
                reverseKernel<<<blocks, threads>>>(reverse, Pr, Pc, Sr, Sc, augment, workers);
                hipDeviceSynchronize();
                augmentKernel<<<blocks, threads>>>(augment, Sr, Sc, Ar, Ac, workers);
                hipDeviceSynchronize();
                hipMemset(Vc, 0, sizeof(int32_t) * workers);
                hipMemset(Vr, 0, sizeof(int32_t) * workers);
                initialize<<<blocks, threads>>>(Pc, -1, workers);
                initialize<<<blocks, threads>>>(Pr, -1, workers);
                initialize<<<blocks, threads>>>(Sr, -1, workers);
                initialize<<<blocks, threads>>>(Sc, -1, workers);
                hipDeviceSynchronize();
                optimalityCheck(countsSum, newCountSums, Ar, Vr, slacks, matchCount, workers);
                initialize<<<blocks, threads>>>(slacks, INT_MAX, workers);
                hipDeviceSynchronize();
                if (*matchCount == workers) {
                    break;
                } else {
                    getMinSlack(slacks, newSlack, newSlack2, workers, thetaCopy);
                    hipMemset(countNewFrontier, 0, sizeof(int32_t));
                    hipDeviceSynchronize();
                    if (flag == 1){
                        dualUpdate<<<blocks, threads>>>(*thetaCopy, Vr, Vc, Pc, Dr, Dc, slacks, workers, newFrontier, countNewFrontier);
                        hipDeviceSynchronize();
                        copyKernel<<<blocks, threads>>>(frontier, newFrontier, workers);
                        hipDeviceSynchronize();
                    } else {
                        dualUpdate<<<blocks, threads>>>(*thetaCopy, Vr, Vc, Pc, Dr, Dc, slacks, workers, frontier, countNewFrontier);
                        hipDeviceSynchronize();
                    }
                    hipMemcpy(lengthOfFrontier_h, countNewFrontier, sizeof(int32_t), hipMemcpyDeviceToHost);
                }
                hipMemset(augment, 0, sizeof(int32_t) * workers);
                hipMemset(reverse, 0, sizeof(int32_t) * workers);
                hipDeviceSynchronize();
            } while(1);
        }
    } while(1);

      hipFree(costsRowwise_d);
      hipFree(costsColwise_d);
      hipFree(Dr);
      hipFree(Dc);
      hipFree(Vr);
      hipFree(Vc);
      hipFree(Ar);
      hipFree(Ac);
      hipFree(Pc);
      hipFree(Pr);
      hipFree(Sc);
      hipFree(Sr);
      hipFree(slacks);
      hipFree(newSlack);
      hipFree(newSlack2);
      hipFree(frontier);
      hipFree(newFrontier);
      hipFree(reverse);
      hipFree(countNewFrontier);
      hipFree(countReverse);
      hipFree(augment);
      hipFree(visited);
      hipFree(countsSum);
      hipFree(newCountSums);
      
      for(int32_t i = 0; i < workers; i++) {
          free(costs[i]);
      }
      printArray(matchCount, 1);
      stopTime(&timer); printf("%f s\n", elapsedTime(timer));
      free(costs);
      free(costsRowwise_h);
      free(costsColwise_h);
      free(matchCount);
      free(lengthOfFrontier_h);
      free(lengthOfReverse_h);
      free(Ar_h);
      free(thetaCopy);
      return 0;
  }
  

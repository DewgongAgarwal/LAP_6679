#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits.h>

__global__ void optimal(int32_t* Ar, int32_t* Vr, int32_t* countsSum, int32_t size, int32_t* Pc, int32_t* Sr, int32_t* Pr, int32_t* Sc, int32_t* Vc, double* slacks) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        Pc[i] = -1;
        Sr[i] = -1;
        Pr[i] = -1;
        Sc[i] = -1;
        Vr[i] = 0;
        Vc[i] = 0;
        slacks[i] = (double) INT_MAX;
        if (Ar[i] != -1) {
            Vr[i] = 1;
            atomicAdd(countsSum, 1);
        }
    }
}
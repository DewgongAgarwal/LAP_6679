#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void forward(int32_t* frontier, int32_t* costs, double* Dr, double* Dc, int32_t* Ac, int32_t* Vr, int32_t* Vc, double* slack, int32_t* Pr, int32_t* Pc, int32_t workers, int32_t* newFrontier, int32_t* reverse, int32_t* countNewFrontier, int32_t* countReverse, int32_t* visited) {
    int32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (j < workers) {
        int32_t inew = Ac[j];
        int32_t cost = 0;
        double slacks = slack[j];
        double DcCopy = Dc[j];
        double DrCopy = 0;
        int32_t vcCopy = Vc[j];
        int32_t inFrontier = 0;
        int32_t inReverse = 0;
        for(int32_t i = 0;  (vcCopy == 0) && (i < workers); i++) {
            if (frontier[i] == 0 || visited[i] == 1) {
                continue;
            }
            DrCopy = Dr[i];
            cost = costs[j * workers + i];
            Pc[j] = (slacks > cost - DrCopy - DcCopy) * i + (slacks <= cost - DrCopy - DcCopy) * Pc[j];
            slacks = (slacks > cost - DrCopy - DcCopy) * (cost - DrCopy - DcCopy) + (slacks <= cost - DrCopy - DcCopy) * slacks;
            inFrontier = inFrontier || (slacks <= 0.0001 && inew != - 1);
            inReverse = inReverse || (slacks <= 0.0001 && inew == - 1);
        }
        slack[j] = slacks;
        if(inFrontier) {
            *countNewFrontier = 1;
            Pr[inew] = j;
            Vc[j] = 1;
            Vr[inew] = 0;
            newFrontier[inew] = 1;
        }
        if(inReverse) {
            *countReverse = 1;
            reverse[j] = 1;
        }
    }
}

__global__ void reverseKernel(int32_t* reverse, int32_t* Pr, int32_t* Pc, int32_t* Sr, int32_t* Sc, int32_t* augment, int32_t size) {
    int32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < size && reverse[j] == 1) {
        int32_t rcur = - 1;
        int32_t ccur = j;
        while (ccur != -1) {
            Sc[ccur] = rcur;
            rcur = Pc[ccur];
            Sr[rcur] = ccur;
            ccur = Pr[rcur];
        }
        augment[rcur] = 1;
    }
}

#include <hip/hip_runtime.h>
#include <stdint.h>

__global__ void augmentInit(int32_t* augment, int32_t size) {
    int32_t j = blockIdx.x * blockDim.x + threadIdx.x;
    if (j < size) {
        augment[j] = 0;
    }
}

__global__ void reverseKernel(int32_t* reverse, int32_t* Pr, int32_t* Pc, 
    int32_t* Sr, int32_t* Sc, int32_t* augment, int32_t size) {
    
    int32_t j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < size && reverse[j] == 1) {
        int32_t rcur = - 1;
        int32_t ccur = j;
        while (ccur != -1) {
            Sc[ccur] = rcur;
            rcur = Pc[ccur];
            Sr[rcur] = ccur;
            ccur = Pr[rcur];
        }
        augment[rcur] = 1;
    }

}

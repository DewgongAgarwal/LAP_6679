#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits.h>

__global__ void findRowMin(int32_t* rowData, double* storage, double* other, int32_t size) {
    int32_t minimum = INT_MAX;
    int32_t rowCopy;
    int32_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for(int32_t i = 0; i < size; i++) {
        if (rowIndex * size + i >= size * size) {
            break;
        }
        rowCopy = rowData[i + rowIndex * size] - other[i];
        minimum = (rowCopy < minimum) * rowCopy + (1 - (rowCopy < minimum)) * minimum;
    }
    if (rowIndex < size) {
        storage[rowIndex] = minimum;
    }
}

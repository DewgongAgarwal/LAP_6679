#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits.h>

__global__ void transpose(int32_t* row, int32_t* col, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size * size) {
        col[(i % size) * size + (i / size)] = row[i];
    }
}

__global__ void findRowMin(int32_t* rowData, double* storage, double* other, int32_t size) {
    int32_t minimum = INT_MAX;
    int32_t rowCopy;
    int32_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (rowIndex < size) {
        for(int32_t i = 0; i < size; i++) {
            rowCopy = rowData[i + rowIndex * size] - other[i];
            minimum = minimum ^ ((rowCopy ^ minimum) & -(rowCopy < minimum));
        }
    }
    if (rowIndex < size) {
        storage[rowIndex] = minimum;
    }
}

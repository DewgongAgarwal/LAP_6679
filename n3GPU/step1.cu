#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rowReduction.cu"

__host__ void step1(int32_t* costsRowwise_d, int32_t* costsColwise_d, int32_t *costsRowwise_h, 
                    double *Dr, double *Dc, int32_t workers, 
                    int32_t blocks, int32_t threads) {

    hipMemcpy(costsRowwise_d, costsRowwise_h, sizeof(int32_t) * workers * workers, hipMemcpyHostToDevice);
    transpose<<<(workers * workers + threads - 1) / threads, threads>>>(costsRowwise_d, costsColwise_d, workers);
    findRowMin<<<blocks, threads>>>(costsRowwise_d, Dr, Dc, workers);
    findRowMin<<<blocks, threads>>>(costsColwise_d, Dc, Dr, workers);

}    
    
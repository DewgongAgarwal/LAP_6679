#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "optimalityCheck.cu"

__host__ bool step2(int32_t *Ar, int32_t *Vr, int32_t *countsSum, int32_t workers, 
                    int32_t *Pc, int32_t *Sr, int32_t *Pr, int32_t *Sc, int32_t *Vc, 
                    double *slacks, int32_t *matchCount, int32_t blocks, int32_t threads){
    

    hipMemset(countsSum, 0, sizeof(int32_t));
    optimal<<<blocks, threads>>>(Ar, Vr, countsSum, workers, Pc, Sr, Pr, Sc, Vc, slacks);
    hipMemcpy(matchCount, countsSum, sizeof(int32_t), hipMemcpyDeviceToHost);
    return (*matchCount == workers);
    
}
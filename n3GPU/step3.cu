#include <hip/hip_runtime.h>
#include <stdbool.h>
#include "forwardPassKernel.cu"

__host__ void step3(int32_t* frontier, int32_t* costs, double* Dr, double* Dc, 
    int32_t* Ar, int32_t* Ac, int32_t* Vr, int32_t* Vc, double* slack, 
    int32_t* Pr, int32_t* Pc, int32_t workers, int32_t* newFrontier, int32_t* repeat,
    int32_t* reverse, int32_t* countNewFrontier, int32_t* countReverse, 
    int32_t* visited, int32_t* goStep4, int32_t blocks, int32_t threads,
    int32_t requiresInitialization, int32_t* lengthOfFrontier, int32_t* lengthOfNewFrontier) {

    hipMemset(lengthOfNewFrontier, 0, sizeof(int32_t));
    if (requiresInitialization)
        hipMemset(lengthOfFrontier, 0, sizeof(int32_t));
        hipMemset(visited, 0, workers * sizeof(int32_t));
        hipMemset(reverse, 0, workers * sizeof(int32_t));
        frontierInit<<<blocks, threads>>>(frontier, newFrontier, visited, 
            reverse, Ar, workers, lengthOfFrontier);

    *repeat = 0;
    int32_t count = 0;
    do {
        if (count == 0) {
            hipMemset(lengthOfNewFrontier, 0, sizeof(int32_t));
            forward<<<blocks, threads>>>(frontier, costs, Dr, Dc, Ac, 
                Vr, Vc, slack, Pr, Pc, workers, newFrontier, reverse, 
                countReverse, visited, lengthOfFrontier, 
                lengthOfNewFrontier);
            hipMemcpy(repeat, lengthOfNewFrontier, sizeof(int32_t), hipMemcpyDeviceToHost);
            visitedInit<<<blocks, threads>>>(visited, frontier, workers, lengthOfFrontier);
            count = 1;
        } else {
            hipMemset(lengthOfFrontier, 0, sizeof(int32_t));
            forward<<<blocks, threads>>>(newFrontier, costs, Dr, Dc, Ac, 
                Vr, Vc, slack, Pr, Pc, workers, frontier, reverse, 
                countReverse, visited, lengthOfNewFrontier, 
                lengthOfFrontier);
            hipMemcpy(repeat, lengthOfFrontier, sizeof(int32_t), hipMemcpyDeviceToHost);
            visitedInit<<<blocks, threads>>>(visited, newFrontier, workers, lengthOfNewFrontier);
            count = 0;
        }
        
    } while(*repeat > 0);
    hipMemcpy(goStep4, countReverse, sizeof(int32_t), hipMemcpyDeviceToHost);
    hipMemset(countReverse, 0, sizeof(int32_t));

}
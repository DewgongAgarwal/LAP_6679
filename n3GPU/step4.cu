#include <hip/hip_runtime.h>
#include "reversePassKernel.cu"

__host__ void step4(int32_t* reverse, int32_t* Pr, int32_t* Pc, int32_t* Sr, int32_t* Sc, 
    int32_t* augment, int32_t workers, int32_t blocks, int32_t threads) {
    // augmentInit<<<blocks, threads>>>(augment, workers);
    hipMemset(augment, 0, workers * sizeof(int32_t));
    reverseKernel<<<blocks, threads>>>(reverse, Pr, Pc, Sr, Sc, augment, workers);

}
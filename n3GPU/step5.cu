#include <hip/hip_runtime.h>
#include "augmentKernel.cu"

__host__ void step5(int32_t* aug, int32_t* Sr, int32_t* Sc, int32_t* Ar, 
    int32_t* Ac, int32_t workers, int32_t blocks, int32_t threads) {

    augmentKernel<<<blocks, threads>>>(aug, Sr, Sc, Ar, Ac, workers);

}
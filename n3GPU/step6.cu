#include <hip/hip_runtime.h>
#include "dualUpdate.cu"

__host__ void getMinSlack(double* slack, double* newSlack, double* newSlack2, 
    int32_t workers, double* minSlack) {
    
    int32_t threads = 512;
    int32_t blocks = (workers + threads - 1) / threads;
    int32_t flag = -1;
    int32_t lengthOfCountSum = workers;
    minKernel<<<blocks, threads>>>(slack, newSlack, lengthOfCountSum);
    lengthOfCountSum = blocks;
    while(lengthOfCountSum > 1) {
        blocks = (lengthOfCountSum + threads - 1) / threads;
        minKernel<<<blocks, threads>>>(newSlack, newSlack2, lengthOfCountSum);
        lengthOfCountSum /= threads;
        flag = 0;
        if (lengthOfCountSum > 1) {
            blocks = (lengthOfCountSum + threads - 1) / threads;
            minKernel<<<blocks, threads>>>(newSlack2, newSlack, lengthOfCountSum);
            lengthOfCountSum /= threads;
            flag = 1;
        }
    }
    if (flag * flag == 1) {
        hipMemcpy(minSlack, newSlack, sizeof(double), hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(minSlack, newSlack2, sizeof(double), hipMemcpyDeviceToHost);
    }

}

__host__ void step6(double* slack, double* newSlack, double* newSlack2, 
    int32_t workers, int32_t* Vr, int32_t* Vc, int32_t* Pc, double* Dr, 
    double* Dc, int32_t* frontier, int32_t* countNewFrontier, int32_t blocks, 
    int32_t threads, int32_t* lengthOfFrontier) {

    double* theta = (double*)calloc(1, sizeof(double));
    getMinSlack(slack, newSlack, newSlack2, workers, theta);
    hipMemset(lengthOfFrontier, 0, sizeof(int32_t));
    dualUpdate<<<blocks, threads>>>(*theta, Vr, Vc, Pc, Dr, Dc, slack, workers, 
        frontier, countNewFrontier, lengthOfFrontier);
    free(theta);

}
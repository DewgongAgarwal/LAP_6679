#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits.h>

__global__ void transpose(int32_t* rows, int32_t* cols, int32_t workers) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < workers * workers) {
        cols[i] = rows[(i % workers) * workers + i / workers];
    }
}

__global__ void findRowMin(int32_t* rowData, int32_t* storage, int32_t* other, int32_t size) {
    int32_t minimum = INT_MAX;
    int32_t rowCopy;
    int32_t rowIndex = blockIdx.x * blockDim.x + threadIdx.x;
    for(int32_t i = 0; i < size; i++) {
        if (rowIndex * size + i >= size * size) {
            break;
        }
        rowCopy = rowData[i + rowIndex * size] - other[i];
        minimum = minimum ^ ((rowCopy ^ minimum) & -(rowCopy < minimum));
    }
    if (rowIndex < size) {
        storage[rowIndex] = minimum;
    }
}

__global__ void subtractFromRow(int32_t* slacks, int32_t* rowMins, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t row = i % size;
    if (i < size * size) {
        slacks[i] = slacks[i] - rowMins[row];
    }
}

__global__ void subtractFromCol(int32_t* slacks, int32_t* colMins, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t col = i / size;
    if (i < size * size) {
        slacks[i] = slacks[i] - colMins[col];
    }
}

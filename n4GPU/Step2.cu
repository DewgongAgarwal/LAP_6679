#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <math.h>

__global__ void initialMatching(int32_t *rowOfStarAtCol, int32_t *colOfStarAtRow, int32_t *coveredRows, int32_t *coveredCols, int32_t *zeroes, int32_t *zeroesSizes, int32_t* repeatKernel, int32_t numberOfDataPerBlock, int32_t workers) {

    __shared__ int32_t repeat, kernelRepeat;
    int32_t thread = threadIdx.x;
    int32_t block = blockIdx.x;
    int32_t blockSize = blockDim.x;

    if (thread == 0) {
        kernelRepeat = 0;
    }

    int32_t numberOfZeros = zeroesSizes[block];

    int32_t zero = -1, zeroRow = -1, zeroCol = -1;

    
    do {
        __syncthreads();
        if (thread == 0) repeat = 0;
        __syncthreads();
        for (int32_t i = thread; i < numberOfZeros; i += blockSize) {
            zero = zeroes[block * numberOfDataPerBlock + i];
            zeroRow = zero % workers;
            zeroCol = zero / workers;
            if(coveredRows[zeroRow] == 0 && coveredCols[zeroCol] == 0) {
                if (!atomicExch((int *)(&coveredRows[zeroRow]),1)) {
                    if(!atomicExch((int*)(&coveredCols[zeroCol]),1)) {
                        rowOfStarAtCol[zeroCol] = zeroRow;
                        colOfStarAtRow[zeroRow] = zeroCol;
                    } else {
                        coveredRows[zeroRow] = 0;
                        repeat = 1;
                        kernelRepeat = 1;
                    }
                }       
            }
        }
        __syncthreads();
    } while (repeat);

    if (kernelRepeat != 0) {
        *repeatKernel = kernelRepeat;
    } else {
        *repeatKernel = 0;
    }
}
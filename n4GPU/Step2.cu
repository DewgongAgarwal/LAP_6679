#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>
#include <math.h>

__global__ void initialMatching(int32_t *rowOfStarAtCol, int32_t *colOfStarAtRow, int32_t *coveredRows, int32_t *coveredCols, int32_t *zeroes, int32_t *zeroesSizes, int32_t* repeatKernel, int32_t numberOfDataPerBlock, int32_t workers) {

    int32_t repeat = 0, kernelRepeat = 0;
    int32_t thread = threadIdx.x;
    int32_t block = blockIdx.x;
    int32_t blockSize = blockDim.x;

    int32_t numberOfZeros = zeroesSizes[block];
    int32_t numberOfZeroesForThisThread = numberOfZeros / blockSize;
    int32_t numberOfZerosUnderMultiple = numberOfZeros % blockSize;
    int32_t startingIndex = 0;
    if (thread < numberOfZerosUnderMultiple) {
        numberOfZeroesForThisThread++;
        startingIndex = thread * numberOfZeroesForThisThread;
    } else {
        startingIndex = numberOfZerosUnderMultiple * (numberOfZeroesForThisThread + 1) + (thread - numberOfZerosUnderMultiple) * numberOfZeroesForThisThread;
    }

    int32_t zero = -1, zeroRow = -1, zeroCol = -1;

    
    do {
        repeat = 0;
        for (int32_t i = 0; i < numberOfZeroesForThisThread && startingIndex + i < numberOfZeros; ++i) {
            zero = zeroes[block * numberOfDataPerBlock + startingIndex + i];
            zeroRow = zero % workers;
            zeroCol = zero / workers;
            if(coveredRows[zeroRow] == 0 && coveredCols[zeroCol] == 0) {
                if (!atomicExch((int *)(&coveredRows[zeroRow]),1)) {
                    if(!atomicExch((int*)(&coveredCols[zeroCol]),1)) {
                        rowOfStarAtCol[zeroCol] = zeroRow;
                        colOfStarAtRow[zeroRow] = zeroCol;
                    } else {
                        coveredRows[zeroRow] = 0;
                        repeat = 1;
                        kernelRepeat = 1;
                    }
                }       
            }
        }
        __syncthreads();
    } while (repeat);

    if (kernelRepeat != 0) {
        *repeatKernel = kernelRepeat;
    } else {
        *repeatKernel = 0;
    }
}

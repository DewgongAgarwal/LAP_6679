#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

//Step 3

__global__ void searchInitialization(int32_t* coveredRows, int32_t* coveredCols, int32_t* numStarredZeroes, int32_t size) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) {
        coveredRows[i] = 0;
        coveredCols[i] = 0;
    }
}

__global__ void searchTermination(int32_t* rowOfStarAtColumn, int32_t* coveredCols, int32_t* numStarredZeroes, int32_t size) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size && rowOfStarAtColumn[i] >= 0) {
        coveredCols[i] = 1;
        atomicAdd(numStarredZeroes, 1);
    }
}
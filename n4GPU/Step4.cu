#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>


__global__ void alternatingGraphPathSearch(int32_t* coveredRows, int32_t* coveredCols, int32_t* zeroes, int32_t* zeroesSizes, int32_t numberOfDataPerBlock, int32_t workers, int32_t* repeatKernel, int32_t* goToStep5, int32_t* colOfStarAtRow, int32_t* colOfPrimeAtRow) {
    volatile int32_t* coveredCols1 = coveredCols;
    volatile int32_t* coveredRows1 = coveredRows;

    __shared__ int32_t zeroFound, kernelRepeat, goToStep5_shared;

    int32_t thread = threadIdx.x;
    int32_t block = blockIdx.x;
    int32_t numberOfZeros = zeroesSizes[block];

    int32_t zero = -1, zeroRow = -1, zeroCol = -1, starCol = -1;

    if (thread == 0) {
        kernelRepeat = 0;
        goToStep5_shared = 0;
    }

    do {
        __syncthreads();
        if (thread == 0) zeroFound = 0;
        __syncthreads();

        for (int32_t i = thread; i < numberOfZeros; i += blockDim.x) {
            zero = zeroes[block * numberOfDataPerBlock + i];
            zeroRow = zero % workers;
            zeroCol = zero / workers;
            starCol = colOfStarAtRow[zeroRow];
            if (coveredCols1[zeroCol] == 0 && coveredRows1[zeroRow] == 0) {
                colOfPrimeAtRow[zeroRow] = zeroCol;
                zeroFound = 1;
                kernelRepeat = 1;
                if (starCol != -1) {
                    coveredRows1[zeroRow] = 1;
                    __threadfence();
                    coveredCols1[starCol] = 0;
                } else {
                    goToStep5_shared = 1;
                }
            }
        }
        __syncthreads();
    } while(zeroFound == 1 && goToStep5_shared == 0);

    if (thread == 0) {
        if (kernelRepeat) *repeatKernel = 1;
	    if (goToStep5_shared) *goToStep5 = 1;
    }

}
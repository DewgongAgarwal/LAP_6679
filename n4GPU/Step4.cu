#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>


__global__ void alternatingGraphPathSearch(int32_t* coveredRows, int32_t* coveredCols, int32_t* zeroes, int32_t* zeroesSizes, int32_t numberOfDataPerBlock, int32_t workers, int32_t* repeatKernel, int32_t* goToStep5, int32_t* colOfStarAtRow, int32_t* colOfPrimeAtRow) {
    volatile int32_t* coveredCols1 = coveredCols;
    volatile int32_t* coveredRows1 = coveredRows;

    __shared__ int32_t zeroFound, goToStep5_shared;
    int32_t kernelRepeat = 0;

    int32_t thread = threadIdx.x;
    int32_t block = blockIdx.x;
    int32_t blockSize = blockDim.x;
    int32_t numberOfZeros = zeroesSizes[block];
    int32_t numberOfZeroesForThisThread = numberOfZeros / blockSize;
    int32_t numberOfZerosUnderMultiple = numberOfZeros % blockSize;
    int32_t startingIndex = 0;
    if (thread < numberOfZerosUnderMultiple) {
        numberOfZeroesForThisThread++;
        startingIndex = thread * numberOfZeroesForThisThread;
    } else {
        startingIndex = numberOfZerosUnderMultiple * (numberOfZeroesForThisThread + 1) + (thread - numberOfZerosUnderMultiple) * numberOfZeroesForThisThread;
    }

    int32_t zero = -1, zeroRow = -1, zeroCol = -1, starCol = -1;

    if (thread == 0) {
        goToStep5_shared = 0;
    }

    do {
        __syncthreads();
        if (thread == 0) zeroFound = 0;
        __syncthreads();

        for (int32_t i = 0; i < numberOfZeroesForThisThread && startingIndex + i < numberOfZeros; ++i) {
            zero = zeroes[block * numberOfDataPerBlock + startingIndex + i];
            zeroRow = zero % workers;
            zeroCol = zero / workers;
            starCol = colOfStarAtRow[zeroRow];
            if (coveredCols1[zeroCol] == 0 && coveredRows1[zeroRow] == 0) {
                colOfPrimeAtRow[zeroRow] = zeroCol;
                zeroFound = 1;
                kernelRepeat = 1;
                if (starCol != -1) {
                    coveredRows1[zeroRow] = 1;
                    __threadfence();
                    coveredCols1[starCol] = 0;
                } else {
                    goToStep5_shared = 1;
                }
            }
        }
        __syncthreads();
    } while(zeroFound == 1 && goToStep5_shared == 0);

    if (thread == 0) {
        if (goToStep5_shared) *goToStep5 = 1;
    }
    if (kernelRepeat) *repeatKernel = 1;
}

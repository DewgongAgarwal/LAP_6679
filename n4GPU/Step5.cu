#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdbool.h>

__global__ void removeOverlappingPaths(int32_t* colOfPrimeAtRow, int32_t* colOfStarAtRow, int32_t* rowOfStarAtCol, int32_t* rowOfGreenAtCol, int32_t size) {
    int32_t i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        int32_t colOfPrime = colOfPrimeAtRow[i];
        int32_t row = 0;
        if (colOfPrime != -1 && colOfStarAtRow[i] == -1) {
            rowOfGreenAtCol[colOfPrime] = i;
            row = rowOfStarAtCol[colOfPrime];
            while (row != -1) {
                colOfPrime = colOfPrimeAtRow[row];
                rowOfGreenAtCol[colOfPrime] = row;
                row = rowOfStarAtCol[colOfPrime];
            }
        }
    }
}

__global__ void alternatingPaths(int32_t* colOfStarAtRow, int32_t* rowOfStarAtCol, int32_t* rowOfGreenAtCol, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int32_t rowZ0 = rowOfGreenAtCol[i];
        int32_t colZ2, colZ0;
        if(rowZ0 != -1 && rowOfStarAtCol[i] == -1) {
            colZ2 = colOfStarAtRow[rowZ0];
            colOfStarAtRow[rowZ0] = i;
            rowOfStarAtCol[i] = rowZ0;
            while(colZ2 != -1) {
                rowZ0 = rowOfGreenAtCol[colZ2];
                colZ0 = colZ2;
                colZ2 = colOfStarAtRow[rowZ0];
                colOfStarAtRow[rowZ0] = colZ0;
                rowOfStarAtCol[colZ0] = rowZ0;
            }
        }
    }
}
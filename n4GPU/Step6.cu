#include <stdint.h>
#include <hip/hip_runtime.h>
#include <limits.h>
#include <math.h>

__global__ void minKernelFirst(int32_t* min, int32_t* newMin, int32_t size, int32_t workers, int32_t* coveredCols, int32_t* coveredRows) {
    __shared__ int32_t partial[512];
    int32_t index = threadIdx.x;
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t col = i / workers;
    int32_t row = i % workers;
    
    int32_t x = 0;
    int32_t y = 0;
    int32_t temp = 0;
    if (i < size) {
        if (coveredCols[col] == 0 && coveredRows[row] == 0) {
            temp = min[i];
        } else {
            temp = INT_MAX;
        }
        partial[threadIdx.x] = (temp > 0) * temp + (1 - temp > 0) * INT_MAX; 
    } else {
        partial[threadIdx.x] = INT_MAX;
    }
    for(int32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();

        if (index < stride) {
            x = partial[index];
            y = partial[index + stride];
            partial[index] = y ^ ((x ^ y) & -(x < y));
        }
    }
    if(index == 0) {
        newMin[blockIdx.x] = partial[index];
    }
}

__global__ void minKernel(int32_t* min, int32_t* newMin, int32_t size) {
    __shared__ int32_t partial[256];
    int32_t index = threadIdx.x;
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t x = 0;
    int32_t y = 0;
    if (i < size) {
        int32_t temp = min[i];
        partial[threadIdx.x] = (temp > 0) * temp + (1 - temp > 0) * INT_MAX; 
    } else {
        int32_t temp = min[0];
        partial[threadIdx.x] = (temp > 0) * temp + (1 - temp > 0) * INT_MAX;
    }
    for(int32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();

        if (index < stride) {
            x = partial[index];
            y = partial[index + stride];
            partial[index] = y ^ ((x ^ y) & -(x < y));
        }
    }
    if(index == 0) {
        newMin[blockIdx.x] = partial[index];
    }
}

__host__ void getMinSlack(int32_t* slack, int32_t* newSlack, int32_t* newSlack2, int32_t workers, int32_t n, int32_t* minSlack, int32_t* coveredCols, int32_t* coveredRows) {
    int32_t threads = 256;
    int32_t threadsBig = 512;
    int32_t blocks = (workers + threadsBig - 1) / threadsBig;
    int32_t flag = -1;
    int32_t lengthOfCountSum = workers;

    minKernelFirst<<<blocks, threadsBig>>>(slack, newSlack, lengthOfCountSum, n, coveredCols, coveredRows);
    hipDeviceSynchronize();
    lengthOfCountSum = blocks;
    while(lengthOfCountSum > 1) {
        blocks = (lengthOfCountSum + threads - 1) / threads;
        minKernel<<<blocks, threads>>>(newSlack, newSlack2, lengthOfCountSum);
        lengthOfCountSum /= threads;
        flag = 0;
        if (lengthOfCountSum > 1) {
            blocks = (lengthOfCountSum + threads - 1) / threads;
            minKernel<<<blocks, threads>>>(newSlack2, newSlack, lengthOfCountSum);
            lengthOfCountSum /= threads;
            flag = 1;
        }
    }

    if (flag * flag == 1) {
        hipMemcpy(minSlack, newSlack, sizeof(int32_t), hipMemcpyDeviceToHost);
    } else {
        hipMemcpy(minSlack, newSlack2, sizeof(int32_t), hipMemcpyDeviceToHost);
    }
}


__global__ void addSub(int32_t min, int32_t blocksStep4, int32_t* zeroesSizes, int32_t* allZeroesSize, int32_t workers, int32_t* slacks, int32_t* coveredRows, int32_t* coveredCols, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int32_t row = i % workers;
        int32_t col = i / workers;
        int32_t covCol = coveredCols[col];
        int32_t covRow = coveredRows[row];
        if (covCol == 0 && covRow == 0) {
            slacks[i] -= min;
        } else if (covCol == 1 && covRow == 1) {
            slacks[i] += min;
        }
    }
    if (i == 0) *allZeroesSize = 0;
	if (i < blocksStep4) zeroesSizes[i] = 0;
}


__global__ void compressKernel(int32_t* slacks, int32_t* zeroes, int32_t* zeroesSizes, int32_t* allZeroesSize, int numberOfDataPerBlock, int32_t workers, int32_t size) {
    
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t offset = -1;
    if (i < size && slacks[i] == 0) {
        int32_t b = i / numberOfDataPerBlock;
        atomicAdd(allZeroesSize, 1);
        offset = atomicAdd(&(zeroesSizes[b]), 1);
        zeroes[b * numberOfDataPerBlock + offset] = i;
    }
}
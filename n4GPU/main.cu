#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include "Step1.cu"
#include "Step2.cu"
#include "Step3.cu"
#include "Step4.cu"
#include "Step5.cu"
#include "Step6.cu"

typedef struct {
    struct timeval startTime;
    struct timeval endTime;
} Timer;

int32_t threads = 64;
int32_t bigThreads = 512;

Timer timer;
Timer timer1;
Timer timer2;
Timer timer3;
Timer timer4;
Timer timer5;
Timer timer6;
float step1Time = 0.0;
float step2Time = 0.0;
float step3Time = 0.0;
float step4Time = 0.0;
float step5Time = 0.0;
float step6Time = 0.0;


hipError_t cuda_ret;

#define FATAL(_msg, ...) \
    do {\
      fprintf(stderr, "[%s:%d] " _msg "\n", __FILE__, __LINE__, ##__VA_ARGS__); \
      exit(-1);								\
    } while(0)

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

__host__ void printArray(int32_t* a, int32_t size) {
      for(int32_t i = 0; i < size; i++) {
          printf("%d, ", a[i]);
      }
      printf("\n");
  }

__host__ void printMatrix(int32_t** a, int32_t size) {
    for(int32_t i = 0; i < size; i++) {
        for(int32_t j = 0; j < size; j++) {
            printf("%d, ", a[i][j]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__ void initialize(int32_t* a, int32_t initialValue, int32_t size) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < size) {
        a[i] = initialValue;
    }
}

  __host__ void flatten(int32_t** costs, int32_t workers, int32_t* costsRowwise) {
    for(int32_t i = 0; i < workers; i++) {
        for(int32_t j = 0; j < workers; j++) {
            costsRowwise[i * workers + j] = costs[i][j];
        }
    }
}

__host__ void unflatten(int32_t** costs, int32_t* cols, int32_t workers) {
    for(int32_t i = 0; i < workers; i++) {
        for(int32_t j = 0; j < workers; j++) {
            costs[j][i] = cols[i * workers + j];
        }
    }
}

__host__ void generateCosts(int32_t** costs, int32_t workers) {
    for(int32_t i = 0; i < workers; i++) {
        for(int32_t j = 0; j < workers; j++) {
            costs[i][j] = rand() % (workers * 10 + 1);
        }
    }
}

__host__ void declare(int32_t** a, int32_t size) {
    cuda_ret = hipMalloc((void**) a, sizeof(int32_t) * size);
    if(cuda_ret != hipSuccess) FATAL("Unable to allocate device memory");
}

int n4Hungarian(int32_t workers) {
    srand(0);
    printf("Workers: %d\n", workers);
    // int32_t array[16] = {9, 22, 58, 11, 43, 78, 72, 50, 41, 28, 91, 37, 74, 42, 27, 49};
    int32_t** costs = (int32_t**)calloc(workers, sizeof(int32_t*));
    for(int32_t i = 0; i < workers; i++) {
        costs[i] = (int32_t*)calloc(workers, sizeof(int32_t));
        // for (int32_t j = 0; j < workers; j++) {
        //     costs[i][j] = array[i * workers + j];
        // }
      }
      generateCosts(costs, workers);
    startTime(&timer);
    int32_t* costsRowwise_h = (int32_t*)calloc(workers * workers, sizeof(int32_t));
    int32_t* repeat = (int32_t*)calloc(1, sizeof(int32_t));
    int32_t* goToStep5 = (int32_t*)calloc(1, sizeof(int32_t));
    int32_t* numStarredZeroes_h = (int32_t*)calloc(1, sizeof(int32_t));
    int32_t* min = (int32_t*)calloc(1, sizeof(int32_t));
    int32_t* answers = (int32_t*)calloc(workers, sizeof(int32_t));
    int32_t* zeroesCopy = (int32_t*)calloc(1, sizeof(int32_t));

    flatten(costs, workers, costsRowwise_h);
    int32_t blocks = (workers + threads - 1) / threads;
    int32_t bigBlocks = (workers * workers + bigThreads - 1) / bigThreads;
    int32_t colPerBlock = 512;
    int32_t newSlackSize = (workers * workers + 512 - 1) / 512;
    int32_t newSlack2Size = (newSlackSize + 256 - 1) / 256;
    int32_t blocksStep2And4 = (workers + colPerBlock - 1) / colPerBlock;

    int32_t* rowMins_d;
    int32_t* colMins_d;
    int32_t* costsRowwise_d;
    int32_t* slacks;
    int32_t* newSlack;
    int32_t* newSlack2;
    int32_t* zeroes;
    int32_t* zeroesSizes_d;
    int32_t* allZeroesSize;
    int32_t* repeatKernel;
    int32_t* rowOfStarAtCol;
    int32_t* colOfStarAtRow;
    int32_t* coveredRows;
    int32_t* coveredCols;
    int32_t* colOfPrimeAtRow;
    int32_t* rowOfGreenAtCol;
    int32_t* numStarredZeroes;
    int32_t* goToStep5Kernel;
    declare(&rowMins_d, workers);
    declare(&colMins_d, workers);
    declare(&costsRowwise_d, workers * workers);
    declare(&slacks, workers * workers);
    declare(&newSlack, newSlackSize);
    declare(&newSlack2, newSlack2Size);
    declare(&zeroes, workers * workers);
    declare(&zeroesSizes_d, blocksStep2And4);
    declare(&allZeroesSize, 1);
    declare(&repeatKernel, 1);
    declare(&rowOfStarAtCol, workers);
    declare(&colOfStarAtRow, workers);
    declare(&coveredRows, workers);
    declare(&coveredCols, workers);
    declare(&colOfPrimeAtRow, workers);
    declare(&rowOfGreenAtCol, workers);
    declare(&numStarredZeroes, 1);
    declare(&goToStep5Kernel, 1);

    hipMemset(allZeroesSize, 0, sizeof(int32_t));
    hipMemset(repeatKernel, 0, sizeof(int32_t));
    hipMemset(goToStep5Kernel, 0, sizeof(int32_t));
    hipMemset(numStarredZeroes, 0, sizeof(int32_t));

    initialize<<<blocks, threads>>>(colMins_d, 0, workers);
    initialize<<<blocks, threads>>>(coveredRows, 0, workers);
    initialize<<<blocks, threads>>>(coveredCols, 0, workers);
    initialize<<<blocks, threads>>>(rowOfStarAtCol, -1, workers);
    initialize<<<blocks, threads>>>(colOfStarAtRow, -1, workers);
    initialize<<<(blocksStep2And4 + threads - 1) / threads, threads>>>(zeroesSizes_d, 0, blocksStep2And4);

    hipMemcpy(costsRowwise_d, costsRowwise_h, sizeof(int32_t) * workers * workers, hipMemcpyHostToDevice);
    transpose<<<bigBlocks, bigThreads>>>(costsRowwise_d, slacks, workers);
    startTime(&timer1);
    findRowMin<<<blocks, threads>>>(costsRowwise_d, rowMins_d, colMins_d, workers);
    
    findRowMin<<<blocks, threads>>>(slacks, colMins_d, rowMins_d, workers);
    

    
    subtractFromRow<<<bigBlocks, bigThreads>>>(slacks, rowMins_d, workers);
    
    subtractFromCol<<<bigBlocks, bigThreads>>>(slacks, colMins_d, workers);
    
    compressKernel<<<bigBlocks, bigThreads>>>(slacks, zeroes, zeroesSizes_d, allZeroesSize, colPerBlock * workers, workers, workers * workers);
    
    stopTime(&timer1);
    step1Time += elapsedTime(timer1);
    startTime(&timer2);
    do {
        *repeat = 0;
        hipMemset(repeatKernel, 0, sizeof(int32_t));
        
        initialMatching<<<blocksStep2And4, colPerBlock>>>(rowOfStarAtCol, colOfStarAtRow, coveredRows, coveredCols, zeroes, zeroesSizes_d, repeatKernel, colPerBlock * workers, workers);
        
        hipMemcpy(repeat, repeatKernel, sizeof(int32_t), hipMemcpyDeviceToHost);
        
    } while(*repeat);
    stopTime(&timer2);
    step2Time += elapsedTime(timer2);

    do {
        startTime(&timer3);
        hipMemset(numStarredZeroes, 0, sizeof(int32_t));
        searchInitialization<<<blocks, threads>>>(coveredRows, coveredCols, numStarredZeroes, workers);
        
        searchTermination<<<blocks, threads>>>(rowOfStarAtCol, coveredCols, numStarredZeroes, workers);
        
        hipMemcpy(numStarredZeroes_h, numStarredZeroes, sizeof(int32_t), hipMemcpyDeviceToHost);
        stopTime(&timer3);
        step3Time += elapsedTime(timer3);
        if (*numStarredZeroes_h == workers) {
            printf("Optimal Solution Found\n");
            hipMemcpy(answers, colOfStarAtRow, sizeof(int32_t) * workers, hipMemcpyDeviceToHost);
            int32_t sum = 0;
            for (int32_t j = 0; j < workers; j++) {
                sum += costs[j][answers[j]];
            }
            printf("Total cost: %d\n", sum);
            break;
        }
        
        initialize<<<blocks, threads>>>(colOfPrimeAtRow, -1, workers);
        initialize<<<blocks, threads>>>(rowOfGreenAtCol, -1, workers);
        

        while(true) {
            startTime(&timer4);
            do {
                hipMemset(repeatKernel, 0, sizeof(int32_t));
                hipMemset(goToStep5Kernel, 0, sizeof(int32_t));
                alternatingGraphPathSearch<<<blocksStep2And4, colPerBlock>>>(coveredRows, coveredCols, zeroes, zeroesSizes_d, colPerBlock * workers, workers, repeatKernel, goToStep5Kernel, colOfStarAtRow, colOfPrimeAtRow);
                
                hipMemcpy(repeat, repeatKernel, sizeof(int32_t), hipMemcpyDeviceToHost);
                hipMemcpy(goToStep5, goToStep5Kernel, sizeof(int32_t), hipMemcpyDeviceToHost);
                
            } while(*repeat && !(*goToStep5));
            stopTime(&timer4);
            step4Time += elapsedTime(timer4);
            if (*goToStep5) {
                break;
            }
            startTime(&timer6);
            getMinSlack(slacks, newSlack, newSlack2, workers * workers, workers, min, coveredCols, coveredRows);
            addSub<<<bigBlocks, bigThreads>>>(*min, blocksStep2And4, zeroesSizes_d, allZeroesSize, workers, slacks, coveredRows, coveredCols, workers * workers);
            
            compressKernel<<<bigBlocks, bigThreads>>>(slacks, zeroes, zeroesSizes_d, allZeroesSize, colPerBlock * workers, workers, workers * workers);
            
            stopTime(&timer6);
            step6Time += elapsedTime(timer6);
        }
        startTime(&timer5);
        removeOverlappingPaths<<<blocks, threads>>>(colOfPrimeAtRow, colOfStarAtRow, rowOfStarAtCol, rowOfGreenAtCol, workers);
        
        alternatingPaths<<<blocks, threads>>>(colOfStarAtRow, rowOfStarAtCol, rowOfGreenAtCol, workers);
        
        stopTime(&timer5);
        step5Time += elapsedTime(timer5);
    } while(true);
    stopTime(&timer); printf("Total Runtime: %f s\n", elapsedTime(timer));
    printf("Step 1 Runtime: %f s\n", step1Time);
    printf("Step 2 Runtime: %f s\n", step2Time);
    printf("Step 3 Runtime: %f s\n", step3Time);
    printf("Step 4 Runtime: %f s\n", step4Time);
    printf("Step 5 Runtime: %f s\n", step5Time);
    printf("Step 6 Runtime: %f s\n", step6Time);
    for(int32_t i = 0; i < workers; i++) {
        free(costs[i]);
    }
    free(costs);
    free(costsRowwise_h);
    free(repeat);
    free(goToStep5);
    free(numStarredZeroes_h);
    free(min);
    free(zeroesCopy);
    free(answers);

    hipFree(rowMins_d);
    hipFree(colMins_d);
    hipFree(costsRowwise_d);
    hipFree(slacks);
    hipFree(newSlack);
    hipFree(newSlack2);
    hipFree(zeroes);
    hipFree(zeroesSizes_d);
    hipFree(allZeroesSize);
    hipFree(repeatKernel);
    hipFree(rowOfStarAtCol);
    hipFree(colOfStarAtRow);
    hipFree(coveredRows);
    hipFree(coveredCols);
    hipFree(colOfPrimeAtRow);
    hipFree(rowOfGreenAtCol);
    hipFree(numStarredZeroes);
    hipFree(goToStep5Kernel);
    return 0;
}

int main() {
    for(int32_t i = 5; i < 15; i++) {
        n4Hungarian(1 << i);
        printf("\n\n");
    }
}

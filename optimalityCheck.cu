#include <hip/hip_runtime.h>
#include <stdint.h>
#include <limits.h>

__global__ void optimal(int32_t* Ar, int32_t* Vr, int32_t* countsSum, int32_t size) {
    __shared__ int32_t count[32];
    int32_t index = threadIdx.x;
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        int32_t copyArr = Ar[i];
        count[threadIdx.x] = copyArr != -1 ? 1 : 0;
        Vr[i] = copyArr != -1 ? 1 : 0; 
    } else {
        count[threadIdx.x] = 0;
    }
    for(int32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (index < stride) {
            count[index] += count[index + stride];
        }
    }
    if(index == 0) {
        countsSum[blockIdx.x] = count[0];
    }
}

__global__ void sumKernel(int32_t* sum, int32_t* newSum, int32_t size) {
    __shared__ int32_t partial[32];
    int32_t index = threadIdx.x;
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        partial[threadIdx.x] = sum[i]; 
    } else {
        partial[threadIdx.x] = 0;
    }
    for(int32_t stride = blockDim.x / 2; stride >= 1; stride /= 2) {
        __syncthreads();
        if (index < stride) {
            partial[index] += partial[index + stride];
        }
    }
    if(index == 0) {
        newSum[blockIdx.x] = partial[0];
    }
}